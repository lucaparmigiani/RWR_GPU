#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>                // std::abs
#include <chrono>
#include <vector>
#include "Timer.cuh"
#include "expMatrix.h"
#include "readExpMatrix.h"
#include "readGraph.h"
#include "cudaExpMatrix.cuh"
#include "test/test.h"

int main(int argc, char* argv[]) {

    if (true) {
    Testing::testRandomEdgeList(F_GRAPH_COO, F_EM);
    }
    else { 
    std::string errString(
	"Syntax Error:\nRWR <graph_path> <expression_matrix_path>\n  or  \nRWR -T      do test on a random graph");

	if (argc < 2){
        error(errString);
    }
    else if (argc < 3){
		std::string parameter = argv[1];
		if (parameter.compare("-T") == 0)
            Testing::multipleTest(Kernel::K3_5);
        else 
            error(errString);

        return 0;
    }
    else if (argc < 4){
        int stop_step = 30;
        double r = 0.6;
        const char* graph_path = argv[1];
        const char* exp_matrix_path = argv[2];
        Testing::testInput(graph_path, exp_matrix_path, stop_step, r);
    }
    }

    //==========
    // TESTING
    //==========
    //BIO
    //Testing::bioTest();
    //Testing::testInput(F_GRAPH_COO, F_EM);
    //ecoli.ppi
    //Testing::testRandom(4787, 300, 169, 0.2);
    //human.ppi
    //Testing::testRandom(19354, 300, 608, 0.2);
    //bio-wormNet
    //Testing::testRandom(16347, 300, 47, 0.2);

    ///* Kernel 1 vs sequential *///
    //std::cout << "Kernel 1 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K1);
    // Kernel 3 vs sequential
    //std::cout << "Kernel 3 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K3);
    // Kernel 3-5 vs sequential
    //std::cout << "Kernel 3+5 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K3_5);
    ///* Kernel 1 vs Kernel 3*///
    //std::cout << "Kernel 3 vs Kernel 1" << '\n';
    //Testing::Kernel1VsKernel3();
    //Testing::Kernel3VsKernel5();
}
