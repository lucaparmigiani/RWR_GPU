#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>                // std::abs
#include <chrono>
#include <vector>
#include "Timer.cuh"
#include "expMatrix.h"
#include "readExpMatrix.h"
#include "readGraph.h"
#include "cudaExpMatrix.cuh"
#include "test/test.h"

int main() {


    //==========
    // TESTING
    //==========
    //BIO
    //Testing::bioTest();
    //Testing::testInput(F_GRAPH_COO, F_EM);
    //ecoli.ppi
    //Testing::testRandom(4787, 300, 169, 0.2);
    //human.ppi
    //Testing::testRandom(19354, 300, 608, 0.2);
    //bio-wormNet
    //Testing::testRandom(16347, 300, 47, 0.2);
    //COG
    Testing::testRandom(308518, 300, 204, 0.2);

    ///* Kernel 1 vs sequential *///
    //std::cout << "Kernel 1 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K1);
    // Kernel 3 vs sequential
    //std::cout << "Kernel 3 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K3);
    // Kernel 3-5 vs sequential
    //std::cout << "Kernel 3+5 vs sequential" << '\n';
    //Testing::multipleTest(Kernel::K3_5);
    ///* Kernel 1 vs Kernel 3*///
    //std::cout << "Kernel 3 vs Kernel 1" << '\n';
    //Testing::Kernel1VsKernel3();
    //Testing::Kernel3VsKernel5();
}
