#pragma once
int num_block;
int BLOCK_DIM;

void cudaExpMatrix::cudaRWR_N(int nof_tests, double r, int stop_step, Kernel k) {
    switch (k) {
        case K1:
		    this->cudaRWR_Kernel1(r, stop_step);
            break;
        case K3:
            this->cudaRWR_Kernel3(r, stop_step);
           break;
        case K3_5:
            hipDeviceProp_t devProp;
            hipGetDeviceProperties(&devProp, 0);
            hipError_t("Device Info");
            size_t sharedMemSizePerBlock = devProp.sharedMemPerBlock;
            size_t sharedMemSizePerSM = devProp.sharedMemPerMultiprocessor;
            size_t numSM = devProp.major;
            const int numPerSM = sharedMemSizePerSM/sharedMemSizePerBlock;

            num_block = numPerSM*numSM;
            BLOCK_DIM = MaxBlockDim;
#if   SMEM_KERNEL==0
            this->cudaRWR_Kernel3(r, stop_step);
            //this->cudaRWR_Kernel1(r, stop_step);
#elif SMEM_KERNEL==1
            if(sharedMemSizePerBlock/(numPerSM*2) - 10000 > this->V)
                this->cudaRWR_Kernel5(r, stop_step);
            else
                this->cudaRWR_Kernel3(r, stop_step);
            //this->cudaRWR_Kernel4(r, stop_step);
#endif
            break;
    }
    hipError_t("RWR Kernel N");

	std::cout	<< std::setprecision(1) << std::fixed << std::endl
				<< "\t    Number of TESTS:  " << nof_tests << std::endl;

	//if (COUNT_DUP && nof_tests == 1) {
	//	int duplicates;
	//	hipMemcpyFromSymbol(&duplicates, HIP_SYMBOL(duplicateCounter), sizeof(int));
	//	std::cout	<< "\t     Duplicates:  " << duplicates << std::endl << std::endl;
	//}
}

void cudaExpMatrix::reset_gpu(){
    hipFree(this->devNodes);
    hipFree(this->devEdges);
    hipFree(this->devDegrees);
	hipFree(this->devExpVec0);
	hipFree(this->devExpVec1);
	hipFree(this->devExpVec2);
}

// ---------------------k AUXILARY FUNCTION ---------------------------------------------

inline void cudaExpMatrix::FrontierDebug(int FrontierSize, bool check) {
    int* F = new int[FrontierSize];
    hipMemcpy( F, devF, FrontierSize * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Frontier content: ";
    for (int i = 0 ; i < FrontierSize ; ++i) {
        std::cout << F[i] << ' ';
    }
    std::cout << '\n';

    if(check) {
        std::sort (F,F+FrontierSize); 
        for (int i = 0 ; i < FrontierSize-1 ; ++i) {
            if(F[i] == F[i+1]){
                std::cout << "Well...that's bad" << '\n';
                return;
            }
        }
    }
}
