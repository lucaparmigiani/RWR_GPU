#include "hip/hip_runtime.h"
#pragma once

__global__ void RWR_init3 ( double* __restrict__ devExpVec0,
                         const double r,
                            const int V ) {
    const int id = Tid + blockIdx.x * blockDim.x;
    if (id < V) {
        devExpVec0[id] *= r;
    }
}

__global__ void RWR_CUDA3 (	int* __restrict__ devNodes,
                            int* __restrict__ devEdges,
                            int* __restrict__ devDegrees,
                         double* __restrict__ devExpVec1,
                         double* __restrict__ devExpVec2,
                         double* __restrict__ devExpVec0,
                                 const double r,
                                    const int V,
                                    const int VWARP_SZ) {

    const int id = Tid + blockIdx.x * blockDim.x;
    const int stride =  blockDim.x * gridDim.x / VWARP_SZ;
    //const int stride =  DIV(V, VWARP_SZ);
    const int vid = id / VWARP_SZ;
    const int mod = Tid & (VWARP_SZ-1);
    for (int k = vid ; k < V ; k+=stride) {
        const int start = devNodes[k];
        const int end = devNodes[k+1];
        double sum = 0;
        devExpVec2[k] = 0;
        for (int i = start + mod ; i < end ; i+=VWARP_SZ) {
            const int e = devEdges[i];
            sum += devExpVec1[e]/devDegrees[e];
        }
        //atomicAdd(devExpVec2 + k, sum);
        atomicAdd(devExpVec2 + k, sum * (1-r));
        //devExpVec2[id] = sum * (1-r) + devExpVec0[id];
        if(mod == 0)
            devExpVec2[k] += devExpVec0[k];
            //devExpVec2[k] *= (1-r);  
    }
//    int id = Tid + blockIdx.x * blockDim.x;
//    if (id < V) {
//        double sum = 0;
//        for (int i = devNodes[id] ; i < devNodes[id+1] ; ++i) {
//            int e = devEdges[i];
//            sum += devExpVec1[e]/devDegrees[e];
//        }
//        devExpVec2[id] = sum * (1-r) + devExpVec0[id];
//    }
}

inline void cudaExpMatrix::cudaRWR_Kernel3(double r, int stop_step) {

    init_kernel3();

    std::cout << " (Kernel 3)" << '\n';
    //int gridDim = min(MAX_CONCURR_TH/BLOCKDIM , DIV(V, BLOCKDIM));
    int gridDim = DIV(V, BLOCKDIM);

    timer::Timer<timer::DEVICE> TM;
    //int mean;
    double time = 0;

    int size = 4; //virtual warp of 4 give best result

    for (int i = 0 ; i < M ; ++i) {
        int level = 0;

        int offset = i*V;
        hipMemcpyAsync((void**) devExpVec0, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec1, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
        hipMemsetAsync((void**) devExpVec2, 0, V*sizeof(double));
  
        TM.start();
        RWR_init3<<<gridDim, BLOCKDIM >>> (devExpVec0, r, V);

        while( level++ < stop_step ) {

            RWR_CUDA3<<<gridDim, BLOCKDIM >>> (devNodes, devEdges, devDegrees, 
                                               devExpVec1, devExpVec2, devExpVec0, 
                                               r, V, size);
            //hipDeviceSynchronize();

            double* tmp = devExpVec1;
            devExpVec1 = devExpVec2;
            devExpVec2 = tmp;

        } 
        TM.stop();
        time += TM.duration();
        hipMemcpyAsync(expMatrix.eMatrix + offset, devExpVec1, V * sizeof(double), hipMemcpyDeviceToHost);
        //std::cout << '\n';
        //for (int j = 0 ; j < V ; ++j) {
        //    std::cout << std::setprecision(5) << expMatrix.eMatrix[j + offset] << ',';
        //}
    }
    std::cout <<"time no data copy: " << time << '\n';
}

void cudaExpMatrix::init_kernel3() {
    hipMalloc(&devNodes,   (V + 1) * sizeof (int));
	hipMalloc(&devEdges,    E * sizeof (int));
	hipMalloc(&devDegrees,  V * sizeof (int));

	hipMemcpy((void**) devNodes,   graph.nodes, (V + 1) * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdges,   graph.edges,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devDegrees, graph.degree, V * sizeof (int), hipMemcpyHostToDevice);

	hipMalloc(&devExpVec0, V * M * sizeof (double));
	hipMalloc(&devExpVec1, V * M * sizeof (double));
	hipMalloc(&devExpVec2, V * M * sizeof (double));

	hipError_t("Graph Allocation");
}

