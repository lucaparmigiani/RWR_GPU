#include "hip/hip_runtime.h"
#pragma once

__global__ void RWR_init6 ( double* __restrict__ devExpVec0,
                                    const double r,
                                       const int V ) {
    const int id = Tid + blockIdx.x * blockDim.x;
    if (id < V) {
        devExpVec0[id] *= r;
    }
}

__global__ void RWR_CUDA6 (	int* __restrict__ devNodes,
                            int* __restrict__ devEdgeU,
                            int* __restrict__ devEdgeV,
                            int* __restrict__ devDegrees,
                         double* __restrict__ devExpVec1,
                         double* __restrict__ devExpVec2,
                         double* __restrict__ devExpVec0,
                                 const double r,
                                    const int E) {

    const int id = Tid + blockIdx.x * blockDim.x;
    //const int stride =  blockDim.x * gridDim.x;
    //for (int k = id ; k < E ; k+=stride) {
    if (id < E) {
        
        const int u = devEdgeU[id];
        const int v = devEdgeV[id];
        const double info_prop = (devExpVec1[u]/devDegrees[u])*(1-r);
        //printf("%d:(%d,%d|%d) ",id, u, v, devNodes[u]);

        atomicAdd(devExpVec2 + v, info_prop);
        //if(id % devNodes[u] == 0 || id == 0 )
        //    devExpVec2[u] += devExpVec0[u];
    }
}

__global__ void RWR_restart( double* __restrict__ devExpVec1,
                             double* __restrict__ devExpVec2,
                             double* __restrict__ devExpVec0,
                                    const int V,
                                    const int level) {
    const int id = Tid + blockIdx.x * blockDim.x;
    if (id < V) {
        devExpVec2[id] += devExpVec0[id];
        //if (level > 1)
        //    devExpVec2[id] -= devExpVec1[id];
    }
}

inline void cudaExpMatrix::cudaRWR_Kernel6(double r, int stop_step) {

    init_kernel6();

    std::cout << " (Kernel 6)" << '\n';
    //int gridDim = min(MAX_CONCURR_TH/BLOCKDIM , DIV(V, BLOCKDIM));
    int gridDimE = DIV(E, BLOCKDIM);
    int gridDimV = DIV(V, BLOCKDIM);

    timer::Timer<timer::DEVICE> TM;
    //int mean;
    double time = 0;

    for (int i = 0 ; i < M ; ++i) {
        int level = 0;

        int offset = i*V;
        hipMemcpyAsync((void**) devExpVec0, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec1, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
  
        TM.start();
        RWR_init6<<<gridDimV, BLOCKDIM >>> (devExpVec0, r, V);

        while( level++ < stop_step ) {

            hipMemsetAsync((void**) devExpVec2, 0, V*sizeof(double));

            RWR_CUDA6<<<gridDimE, BLOCKDIM >>> (devNodes, devEdgeU, devEdgeV, devDegrees, 
                                               devExpVec1, devExpVec2, devExpVec0, 
                                               r, E);
            //hipDeviceSynchronize();
            RWR_restart<<<gridDimV, BLOCKDIM >>>(devExpVec1, devExpVec2, devExpVec0, V, level);


            double* tmp = devExpVec1;
            devExpVec1 = devExpVec2;
            devExpVec2 = tmp;

        } 
        TM.stop();
        time += TM.duration();
        hipMemcpyAsync(expMatrix.eMatrix + offset, devExpVec1, V * sizeof(double), hipMemcpyDeviceToHost);
        //std::cout << '\n';
        //for (int j = 0 ; j < V ; ++j) {
        //    std::cout << std::setprecision(5) << expMatrix.eMatrix[j + offset] << ',';
        //}
    }
    std::cout <<"time no data copy: " << time << '\n';
}

void cudaExpMatrix::init_kernel6() {
    hipMalloc(&devNodes,   (V + 1) * sizeof (int));
	hipMalloc(&devEdgeU,    E * sizeof (int));
	hipMalloc(&devEdgeV,    E * sizeof (int));
	hipMalloc(&devDegrees,  V * sizeof (int));

	hipMemcpy((void**) devNodes,   graph.nodes, (V + 1) * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdgeU,   graph_edgeList.edgeU,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdgeV,   graph_edgeList.edgeV,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devDegrees, graph.degree, V * sizeof (int), hipMemcpyHostToDevice);

	hipMalloc(&devExpVec0, V * M * sizeof (double));
	hipMalloc(&devExpVec1, V * M * sizeof (double));
	hipMalloc(&devExpVec2, V * M * sizeof (double));

	hipError_t("Graph Allocation");
}

