#include "hip/hip_runtime.h"
#pragma once

const int SMemMargin = 10000;
extern int num_block;
extern int BLOCK_DIM;

__global__ void RWR_init5 ( double* __restrict__ devExpVec0,
                         const double r,
                            const int V ) {
    const int id = Tid + blockIdx.x * V;
    const int stride = blockDim.x;
    for (int i = id ; i < V*(blockIdx.x+1) ; i+=stride) {
        devExpVec0[i] *= r;
    }
}

__global__ void RWR_CUDA5 (	int* __restrict__ devNodes,
                            int* __restrict__ devEdges,
                            int* __restrict__ devDegrees,
                         double* __restrict__ devExpVec1,
                         double* __restrict__ devExpVec2,
                         double* __restrict__ devExpVec0,
                                 const double r,
                                    const int V,
                                    const int M,
                                    const int VWARP_SZ) {

    __shared__ double vec1SMem[(SMem_Per_SM/4)-SMemMargin];
    __shared__ double vec2SMem[(SMem_Per_SM/4)-SMemMargin];

    const int vid = (Tid / VWARP_SZ);
    const int stride =  blockDim.x / VWARP_SZ;
    const int mod = Tid & (VWARP_SZ-1);
    const int offset = blockIdx.x*V;

    if (vid + offset < V * M) {
        for (int k = vid , m = vid + offset; k < V ; k+=stride , m+=stride) {
            vec1SMem[k] = devExpVec1[m];
        }
        __syncthreads();


        for (int k = vid , m = vid + offset; k < V ; k+=stride , m+=stride) {
            const int start = devNodes[k];
            const int end = devNodes[k+1];
            double sum = 0;
            vec2SMem[k] = 0;
            for (int i = start + mod ; i < end ; i+=VWARP_SZ) {
                const int e = devEdges[i];
                sum += vec1SMem[e]/devDegrees[e + offset];
            }
            atomicAdd(vec2SMem + k, sum * (1-r));
            if(mod == 0)
                devExpVec2[m] = vec2SMem[k] + devExpVec0[m];
        }
    }
}

inline void cudaExpMatrix::cudaRWR_Kernel5(double r, int stop_step) {

    init_kernel5();

    std::cout << " (Kernel 5)" << '\n';
    //int num_block = min(MAX_CONCURR_TH/BLOCKDIM , DIV(V, BLOCKDIM));
    //int num_block = DIV(V, BLOCKDIM);

    timer::Timer<timer::DEVICE> TM;
    double time = 0;

    //int size = MIN_V(16, 32);
    int size = 4; //virtual warp of 4 give best result

    for (int i = 0 ; i < M ; i+=num_block) {
        int level = 0;

        int offset = i*V;
        int margin = MIN_V(M*V, V*num_block);
        
        hipMemcpyAsync((void**) devExpVec0, expMatrix.eMatrix + offset, margin * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec1, expMatrix.eMatrix + offset, margin * sizeof (double), hipMemcpyHostToDevice);
        hipMemsetAsync((void**) devExpVec2, 0, margin * sizeof(double));
        //hipMemcpyAsync((void**) devExpVec2, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
  
        TM.start();
        RWR_init5<<<num_block, BLOCK_DIM >>> (devExpVec0, r, V);
        hipDeviceSynchronize();

        while( level < stop_step ) {

            RWR_CUDA5<<<num_block, BLOCK_DIM >>> (devNodes, devEdges, devDegrees, 
                                               devExpVec1, devExpVec2, devExpVec0, 
                                               r, V, M, size);
            //hipDeviceSynchronize();

            double* tmp = devExpVec1;
            devExpVec1 = devExpVec2;
            devExpVec2 = tmp;

            level++;
        } 
        //std::cout << i << '\n';
        TM.stop();
        time += TM.duration();
        hipMemcpyAsync(expMatrix.eMatrix + offset, devExpVec1, margin * sizeof(double), hipMemcpyDeviceToHost);
    }
    std::cout <<"time no data copy: " << time << '\n';
}

void cudaExpMatrix::init_kernel5() {
    hipMalloc(&devNodes,   (V + 1) * sizeof (int));
	hipMalloc(&devEdges,    E * sizeof (int));
	hipMalloc(&devDegrees,  V * sizeof (int));

	hipMemcpy((void**) devNodes,   graph.nodes, (V + 1) * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdges,   graph.edges,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devDegrees, graph.degree, V * sizeof (int), hipMemcpyHostToDevice);

    hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	hipError_t("Device Info");
    size_t sharedMemSizePerBlock = devProp.sharedMemPerBlock;
    size_t sharedMemSizePerSM = devProp.sharedMemPerMultiprocessor;
    size_t numSM = devProp.major;
    int numPerSM = sharedMemSizePerSM/sharedMemSizePerBlock;
    num_block = numPerSM*numSM;

    BLOCK_DIM = MaxBlockDim;

	hipMalloc(&devExpVec0, V * M * num_block * sizeof (double));
	hipMalloc(&devExpVec1, V * M * num_block * sizeof (double));
	hipMalloc(&devExpVec2, V * M * num_block * sizeof (double));

	hipError_t("Graph Allocation");
}

