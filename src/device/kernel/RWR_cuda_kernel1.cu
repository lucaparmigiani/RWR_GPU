#include "hip/hip_runtime.h"
#pragma once

__device__ int counter;

__global__
void RWR_init ( double* __restrict__ devExpVec0,
                   int* __restrict__ devF,
                           const int devFSize,
                        const double r,
                               int* devInF) {
    int id = Tid + blockIdx.x * blockDim.x;
    if (id < devFSize) {
        int v = devF[id];
        devInF[v] = 1;
        devExpVec0[v] *= r;
    }
}

__global__
void RWR_CUDA (	int* __restrict__ devNodes,
				int* __restrict__ devEdges,
                int* __restrict__ devDegrees,
                int* __restrict__ devF,
             double* __restrict__ devExpVec1,
             double* __restrict__ devExpVec2,
				        const int devFSize,
                            int* devInF) {

    int id = Tid + blockIdx.x * blockDim.x;
    if (id < devFSize) {
        int localChecker;
        int v = devF[id];
        //devExpVec2[v] -= devExpVec1[v];
        atomicAdd(&devExpVec2[v], -devExpVec1[v]);
        double prop = devExpVec1[v]/devDegrees[v];
        for (int i = devNodes[v] ; i < devNodes[v+1] ; ++i) {
            int e = devEdges[i];
            //devExpVec2[e] += prop;
            atomicAdd(&devExpVec2[e], prop);
            localChecker = *(volatile int*)&devInF[e];
            if (!atomicCAS(&devInF[e], localChecker, 1)) {
                int index = atomicAdd(&counter, 1);
                devF[index] = e;
            }
        }
    }
}

__global__
void RWR_mul ( double* __restrict__ devExpVec0,
               double* __restrict__ devExpVec1,
               double* __restrict__ devExpVec2,
                  int* __restrict__ devF,
                          const int devFSize,
                       const double r) {

    int id = Tid + blockIdx.x * blockDim.x;
    if (id < devFSize) {
        int v = devF[id];
        double tmp = devExpVec2[v]*(1-r) + devExpVec0[v];
        devExpVec2[v] = tmp;
        devExpVec1[v] = tmp;
    }
}

inline void cudaExpMatrix::cudaRWR_Kernel1(double r, int stop_step) {

    std::cout << " (Kernel 1)" << '\n';
    init_kernel1();

    for (int i = 0 ; i < M ; ++i) {

        //Search sources
        int* sources = new int[V];
        int nof_sources = 0;
        for (int j = 0 ; j < V ; j++) {
            double tmp = expMatrix.eMatrix[i*V + j];
            if (tmp != 0) {
                sources[nof_sources++] = j;
            }
        }
        
		reset_kernel1(sources, nof_sources);

        int FrontierSize = nof_sources;
        hipMemcpyToSymbol(HIP_SYMBOL(counter), &FrontierSize, sizeof (int));
        int level = 0;

        int offset = i*V;
        hipMemcpyAsync((void**) devExpVec0, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec1, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec2, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);

  
        if(FrontierSize) {
            //int gridDim = min(MAX_CONCURR_TH/BLOCKDIM , DIV(FrontierSize, BLOCKDIM));
            int gridDim = DIV(FrontierSize, BLOCKDIM);
            RWR_init<<<gridDim, BLOCKDIM >>>
                (devExpVec0, devF, FrontierSize, r, devInF);

            while( level < stop_step ) {

                RWR_CUDA<<<gridDim, BLOCKDIM >>>
                    (devNodes, devEdges, devDegrees, devF, devExpVec1, devExpVec2, FrontierSize, devInF);

                hipMemcpyFromSymbol(&FrontierSize, HIP_SYMBOL(counter), sizeof (int));
                //hipMemcpy(FrontierSize, counter, sizeof (int), hipMemcpyDeviceToHost);
                gridDim = min(MAX_CONCURR_TH/BLOCKDIM , DIV(FrontierSize, BLOCKDIM));


                RWR_mul<<<gridDim, BLOCKDIM >>>
                    (devExpVec0, devExpVec1, devExpVec2, devF, FrontierSize, r);

                level++;
            } 
            hipMemcpyAsync(expMatrix.eMatrix + i*V, devExpVec2, V * sizeof(double), hipMemcpyDeviceToHost);
        }
    }

    hipFree(devF);
    hipFree(devInF);
    //reset_gpu();
}

void cudaExpMatrix::reset_kernel1(const int Sources[], int nof_sources) {
	hipMemcpy(devF, Sources, nof_sources * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&devInF, V*sizeof(int));
    hipMemset(devInF, 0, V*sizeof(int));

	hipError_t("Graph Reset");
}

void cudaExpMatrix::init_kernel1() {
    hipMalloc(&devNodes,   (V + 1) * sizeof (int));
	hipMalloc(&devEdges,    E * sizeof (int));
	hipMalloc(&devDegrees,  V * sizeof (int));

	hipMemcpy((void**) devNodes,   graph.nodes, (V + 1) * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdges,   graph.edges,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devDegrees, graph.degree, V * sizeof (int), hipMemcpyHostToDevice);

	hipMalloc(&devExpVec0, V * sizeof (double));
	hipMalloc(&devExpVec1, V * sizeof (double));
	hipMalloc(&devExpVec2, V * sizeof (double));

	hipError_t("Graph Allocation");
	
	// --------------- Frontier Allocation -------------------

	size_t free, total;
	hipMemGetInfo(&free, &total);
    size_t frontierSize = (free) - 20000 * 1024;

	hipMalloc(&devF, frontierSize);
	allocFrontierSize = frontierSize / sizeof(int);

	hipError_t("Graph Frontier Allocation");
}
