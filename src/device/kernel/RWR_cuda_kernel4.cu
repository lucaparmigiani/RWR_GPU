#include "hip/hip_runtime.h"
#pragma once

extern int num_block;
extern int BLOCK_DIM;

__global__ void RWR_init4 ( double* __restrict__ devExpVec0,
                         const double r,
                            const int V ) {
    const int id = Tid + blockIdx.x * V;
    const int stride = blockDim.x;
    for (int i = id ; i < V*(blockIdx.x+1) ; i+=stride) {
        devExpVec0[i] *= r;
    }
}

__global__ void RWR_CUDA4 (	int* __restrict__ devNodes,
                            int* __restrict__ devEdges,
                            int* __restrict__ devDegrees,
                         double* __restrict__ devExpVec1,
                         double* __restrict__ devExpVec2,
                         double* __restrict__ devExpVec0,
                                 const double r,
                                    const int V,
                                    const int VWARP_SZ) {

    const int stride =  blockDim.x / VWARP_SZ;
    //const int vid = (Tid / VWARP_SZ) + blockIdx.x * V;
    const int vid = (Tid / VWARP_SZ);
    const int mod = Tid & (VWARP_SZ-1);
    const int offset = blockIdx.x*V;

    for (int k = vid , m = vid + offset; k < V ; k+=stride , m+=stride) {
        const int start = devNodes[k];
        const int end = devNodes[k+1];
        double sum = 0;
        devExpVec2[m] = 0;
        for (int i = start + mod ; i < end ; i+=VWARP_SZ) {
            const int e = devEdges[i] + offset;
            sum += devExpVec1[e]/devDegrees[e];
        }
        atomicAdd(devExpVec2 + m, sum * (1-r));
        if(mod == 0)
            devExpVec2[m] += devExpVec0[m];
    }
}

inline void cudaExpMatrix::cudaRWR_Kernel4(double r, int stop_step) {

    init_kernel4();

    std::cout << " (Kernel 4)" << '\n';
    //int num_block = min(MAX_CONCURR_TH/BLOCKDIM , DIV(V, BLOCKDIM));
    //int num_block = DIV(V, BLOCKDIM);

    timer::Timer<timer::DEVICE> TM;
    double time = 0;

    //int size = MIN_V(16, 32);
    int size = 4; //virtual warp of 4 give best result

    for (int i = 0 ; i < M ; i+=num_block) {
        int level = 0;

        int offset = i*V;
        
        hipMemcpyAsync((void**) devExpVec0, expMatrix.eMatrix + offset, V * num_block * sizeof (double), hipMemcpyHostToDevice);
        hipMemcpyAsync((void**) devExpVec1, expMatrix.eMatrix + offset, V * num_block * sizeof (double), hipMemcpyHostToDevice);
        hipMemsetAsync((void**) devExpVec2, 0, V * num_block * sizeof(double));
        //hipMemcpyAsync((void**) devExpVec2, expMatrix.eMatrix + offset, V * sizeof (double), hipMemcpyHostToDevice);
  
        TM.start();
        RWR_init4<<<num_block, BLOCK_DIM >>> (devExpVec0, r, V);
        hipDeviceSynchronize();

        while( level < stop_step ) {

            RWR_CUDA4<<<num_block, BLOCK_DIM >>> (devNodes, devEdges, devDegrees, 
                                               devExpVec1, devExpVec2, devExpVec0, 
                                               r, V, size);
            //hipDeviceSynchronize();

            double* tmp = devExpVec1;
            devExpVec1 = devExpVec2;
            devExpVec2 = tmp;

            level++;
        } 
        //std::cout << i << '\n';
        TM.stop();
        time += TM.duration();
        hipMemcpyAsync(expMatrix.eMatrix + offset, devExpVec1, V * num_block * sizeof(double), hipMemcpyDeviceToHost);
    }
    std::cout <<"time no data copy: " << time << '\n';
}

void cudaExpMatrix::init_kernel4() {
    hipMalloc(&devNodes,   (V + 1) * sizeof (int));
	hipMalloc(&devEdges,    E * sizeof (int));
	hipMalloc(&devDegrees,  V * sizeof (int));

	hipMemcpy((void**) devNodes,   graph.nodes, (V + 1) * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devEdges,   graph.edges,  E * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy((void**) devDegrees, graph.degree, V * sizeof (int), hipMemcpyHostToDevice);

    hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	hipError_t("Device Info");
    size_t sharedMemSizePerBlock = devProp.sharedMemPerBlock;
    size_t sharedMemSizePerSM = devProp.sharedMemPerMultiprocessor;
    size_t numSM = devProp.major;
    int numPerSM = sharedMemSizePerSM/sharedMemSizePerBlock;
    num_block = numPerSM*numSM;

    BLOCK_DIM = MaxBlockDim;

	hipMalloc(&devExpVec0, V * M * num_block * sizeof (double));
	hipMalloc(&devExpVec1, V * M * num_block * sizeof (double));
	hipMalloc(&devExpVec2, V * M * num_block * sizeof (double));

	hipError_t("Graph Allocation");
}

