#include "hip/hip_runtime.h"
#pragma once

const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;

__global__
void matrixTransposeKernelShM(const double* d_matrix_in,
                                    double* d_matrix_out,
                                       int  N, int M ) {

    __shared__ int sh_matrix[BLOCK_SIZE_X*BLOCK_SIZE_Y];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < N && y < M){
        sh_matrix[threadIdx.x +  threadIdx.y * BLOCK_SIZE_Y]= d_matrix_in[x*M + y];

        __syncthreads();
        d_matrix_out[y*N + x] = sh_matrix[threadIdx.x +  threadIdx.y * BLOCK_SIZE_Y];
    }
}

void cudaExpMatrix::transpose(){
    double *d_matrix_out, *d_matrix_in;
    std::cout << "\nTransposing expression matrix...";
    hipMalloc( &d_matrix_in , V*M * sizeof(double));
    hipMalloc( &d_matrix_out, M*V * sizeof(double));

    hipMemcpy( d_matrix_in, expMatrix.eMatrix, V*M * sizeof(double), hipMemcpyHostToDevice);

    hipError_t("Matrix Allocation Trasposition");


    dim3 num_blocks(V/BLOCK_SIZE_X, V/BLOCK_SIZE_Y, 1);
    if (V%BLOCK_SIZE_X) {
        num_blocks.x++;
        num_blocks.y++;
    }
    dim3 block_size(BLOCK_SIZE_X , BLOCK_SIZE_Y, 1);

    matrixTransposeKernelShM<<< num_blocks, block_size >>>(d_matrix_in, d_matrix_out, V, M);

    hipError_t("Matrix Trasposition");
    //double* h_matrix_tmp = new double[M*V];
    hipMemcpy( expMatrix.eMatrix, d_matrix_out, M*V*sizeof(double), hipMemcpyDeviceToHost);

    hipError_t("Matrix Trasposition copy output");

    hipFree( d_matrix_in  );
    hipFree( d_matrix_out );
    std::cout << "Complete" << '\n';

    //for (int i = 0 ; i < M ; ++i) {
    //    for (int j = 0 ; j < V ; ++j) {
    //        std::cout << h_matrix_tmp[i*V + j] << ' ';
    //    }
    //    std::cout << '\n';
    //}
}
